//===--- omptarget.cu - OpenMP GPU initialization ---------------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// This file contains the initialization code for the GPU
//
//===----------------------------------------------------------------------===//
#pragma omp declare target

#include "common/omptarget.h"
#include "common/support.h"
#include "target_impl.h"

////////////////////////////////////////////////////////////////////////////////
// global data tables
////////////////////////////////////////////////////////////////////////////////

extern omptarget_nvptx_Queue<omptarget_nvptx_ThreadPrivateContext,
                             OMP_STATE_COUNT>
    omptarget_nvptx_device_State[MAX_SM];

////////////////////////////////////////////////////////////////////////////////
// init entry points
////////////////////////////////////////////////////////////////////////////////

static void __kmpc_generic_kernel_init() {
  PRINT(LD_IO, "call to __kmpc_kernel_init with version %f\n",
        OMPTARGET_NVPTX_VERSION);

  if (GetLaneId() == 0)
    parallelLevel[GetWarpId()] = 0;

  int threadIdInBlock = __kmpc_get_hardware_thread_id_in_block();
  if (threadIdInBlock != GetMasterThreadID())
    return;

  setExecutionParameters(OMP_TGT_EXEC_MODE_GENERIC, OMP_TGT_RUNTIME_INITIALIZED);
  ASSERT0(LT_FUSSY, threadIdInBlock == GetMasterThreadID(),
          "__kmpc_kernel_init() must be called by team master warp only!");
  PRINT0(LD_IO, "call to __kmpc_kernel_init for master\n");

  // Get a state object from the queue.
  int slot = __kmpc_impl_smid() % MAX_SM;
  usedSlotIdx = slot;
  omptarget_nvptx_threadPrivateContext =
      omptarget_nvptx_device_State[slot].Dequeue();

  // init thread private
  int threadId = 0;
  omptarget_nvptx_threadPrivateContext->InitThreadPrivateContext(threadId);

  // init team context
  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  currTeamDescr.InitTeamDescr();
  // this thread will start execution... has to update its task ICV
  // to point to the level zero task ICV. That ICV was init in
  // InitTeamDescr()
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(
      threadId, currTeamDescr.LevelZeroTaskDescr());

  // set number of threads and thread limit in team to started value
  omptarget_nvptx_TaskDescr *currTaskDescr =
      omptarget_nvptx_threadPrivateContext->GetTopLevelTaskDescr(threadId);
  nThreads = GetNumberOfWorkersInTeam();
  threadLimit = nThreads;

  __kmpc_data_sharing_init_stack();
  __kmpc_impl_target_init();
  omptarget_nvptx_workFn = 0; // Initialized to zero in case there is no work
#ifdef OMPD_SUPPORT
  ompd_init();
  ompd_init_thread_master();
  ompd_bp_thread_begin();
#endif /*OMPD_SUPPORT*/
}

static void __kmpc_generic_kernel_deinit() {
  PRINT0(LD_IO, "call to __kmpc_kernel_deinit\n");
  // Enqueue omp state object for use by another team.
  int slot = usedSlotIdx;
  omptarget_nvptx_device_State[slot].Enqueue(
      omptarget_nvptx_threadPrivateContext);
#ifdef OMPD_SUPPORT
  ompd_bp_thread_end();
#endif
  // Done with work.  Kill the workers.
  omptarget_nvptx_workFn = 0;
#ifdef __AMDGCN__
  omptarget_master_ready = true;
  __kmpc_impl_syncthreads();
#endif
}

static void __kmpc_spmd_kernel_init(bool RequiresFullRuntime) {
  PRINT0(LD_IO, "call to __kmpc_spmd_kernel_init\n");

  setExecutionParameters(OMP_TGT_EXEC_MODE_SPMD,
                         RequiresFullRuntime ? OMP_TGT_RUNTIME_INITIALIZED
                                             : OMP_TGT_RUNTIME_UNINITIALIZED);
  int threadId = __kmpc_get_hardware_thread_id_in_block();
  if (threadId == 0) {
    usedSlotIdx = __kmpc_impl_smid() % MAX_SM;
  }

  __kmpc_impl_syncwarp(__kmpc_impl_activemask());
  if (GetLaneId() == 0) {
    parallelLevel[GetWarpId()] = (__kmpc_get_hardware_num_threads_in_block() > 1
                                      ? OMP_ACTIVE_PARALLEL_LEVEL
                                      : 0);
    __kmpc_impl_threadfence();
  }
  __kmpc_impl_syncwarp(__kmpc_impl_activemask());

  __kmpc_data_sharing_init_stack();
  if (!RequiresFullRuntime)
    return;

  //
  // Team Context Initialization.
  //
  // In SPMD mode there is no master thread so use any cuda thread for team
  // context initialization.
  if (threadId == 0) {
    // Get a state object from the queue.
    omptarget_nvptx_threadPrivateContext =
        omptarget_nvptx_device_State[usedSlotIdx].Dequeue();

    omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
    omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();
    // init team context
    currTeamDescr.InitTeamDescr();
#ifdef OMPD_SUPPORT
    ompd_init();
    ompd_bp_parallel_begin(); // This should be placed later, but the parallel
                              // handle is ready from here on.
#endif /*OMPD_SUPPORT*/
  }
  __kmpc_impl_syncthreads();

  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();

  //
  // Initialize task descr for each thread.
  //
  omptarget_nvptx_TaskDescr *newTaskDescr =
      omptarget_nvptx_threadPrivateContext->Level1TaskDescr(threadId);
  ASSERT0(LT_FUSSY, newTaskDescr, "expected a task descr");
  newTaskDescr->InitLevelOneTaskDescr(currTeamDescr.LevelZeroTaskDescr());
  // install new top descriptor
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(threadId,
                                                             newTaskDescr);

  // init thread private from init value
  int ThreadLimit = GetNumberOfProcsInTeam(/* IsSPMD */ true);
  PRINT(LD_PAR,
        "thread will execute parallel region with id %d in a team of "
        "%d threads\n",
        (int)newTaskDescr->ThreadId(), (int)ThreadLimit);

#ifdef OMPD_SUPPORT
  ompd_init_thread_parallel(); // __kmpc_kernel_parallel() is not called in
                               // spmd mode
  ompd_bp_thread_begin();
#endif
}

static void __kmpc_spmd_kernel_deinit(bool RequiresFullRuntime) {
  // We're not going to pop the task descr stack of each thread since
  // there are no more parallel regions in SPMD mode.
  if (!RequiresFullRuntime)
    return;

  __kmpc_impl_syncthreads();
#ifdef OMPD_SUPPORT
  ompd_bp_thread_end();
#endif
  int threadId = __kmpc_get_hardware_thread_id_in_block();
  if (threadId == 0) {
#ifdef OMPD_SUPPORT
    ompd_bp_parallel_end();
#endif
    // Enqueue omp state object for use by another team.
    int slot = usedSlotIdx;
    omptarget_nvptx_device_State[slot].Enqueue(
        omptarget_nvptx_threadPrivateContext);
  }
}

// Return true if the current target region is executed in SPMD mode.
// NOTE: This function has to return 1 for SPMD mode, and 0 for generic mode.
// That's because `__kmpc_parallel_51` checks if it's already in parallel region
// by comparision between the parallel level and the return value of this
// function.
EXTERN int8_t __kmpc_is_spmd_exec_mode() {
  return (execution_param & OMP_TGT_EXEC_MODE_SPMD) == OMP_TGT_EXEC_MODE_SPMD;
}

EXTERN int8_t __kmpc_is_generic_main_thread(kmp_int32 Tid) {
  return !__kmpc_is_spmd_exec_mode() && __kmpc_is_generic_main_thread_id(Tid);
}

NOINLINE EXTERN int8_t __kmpc_is_generic_main_thread_id(kmp_int32 Tid) {
  return GetMasterThreadID() == Tid;
}

EXTERN
void __kmpc_workers_start_barriers(ident_t *Ident, int TId) {
#ifdef __AMDGCN__
  omptarget_workers_done = true;
  __kmpc_barrier_simple_spmd(Ident, TId);
  while (!omptarget_master_ready)
    __kmpc_barrier_simple_spmd(Ident, TId);

  omptarget_workers_done = false;
#else
  __kmpc_barrier_simple_spmd(Ident, TId);
#endif
}

EXTERN
void __kmpc_workers_done_barriers(ident_t *Ident, int TId) {
  // This worker termination logic permits full barriers in reductions
  // by keeping the master thread waiting at another barrier till
  // all workers are finished.
#ifdef __AMDGCN__
  if (TId == 0)
    omptarget_workers_done = true;
#endif
  __kmpc_barrier_simple_spmd(Ident, TId);
}

EXTERN bool __kmpc_kernel_parallel(void**WorkFn);

static void __kmpc_target_region_state_machine(ident_t *Ident) {

  int TId = __kmpc_get_hardware_thread_id_in_block();
  do {
    void* WorkFn = 0;

    // Wait for the signal that we have a new work function.
    __kmpc_workers_start_barriers(Ident, TId);

    // Retrieve the work function from the runtime.
    bool IsActive = __kmpc_kernel_parallel(&WorkFn);

    // If there is nothing more to do, break out of the state machine by
    // returning to the caller.
    if (!WorkFn)
      return;

    if (IsActive) {
      ((void (*)(uint16_t, uint32_t))WorkFn)(0, TId);
      __kmpc_kernel_end_parallel();
    }

    __kmpc_workers_done_barriers(Ident, TId);

  } while (true);
}

EXTERN
int32_t __kmpc_target_init(ident_t *Ident, int8_t Mode,
                           bool UseGenericStateMachine,
                           bool RequiresFullRuntime) {
#ifdef __AMDGCN__
  omptarget_workers_done = false;
  omptarget_master_ready = false;
#endif

  const bool IsSPMD = Mode & OMP_TGT_EXEC_MODE_SPMD;
  int TId = __kmpc_get_hardware_thread_id_in_block();
  if (IsSPMD)
    __kmpc_spmd_kernel_init(RequiresFullRuntime);
  else
    __kmpc_generic_kernel_init();

   if (IsSPMD) {
    __kmpc_barrier_simple_spmd(Ident, TId);
     return -1;
   }

   if (TId == GetMasterThreadID())
     return -1;

  if (UseGenericStateMachine)
    __kmpc_target_region_state_machine(Ident);

  return TId;
}

EXTERN
void __kmpc_target_deinit(ident_t *Ident, int8_t Mode,
                          bool RequiresFullRuntime) {
  const bool IsSPMD = Mode & OMP_TGT_EXEC_MODE_SPMD;
  if (IsSPMD)
    __kmpc_spmd_kernel_deinit(RequiresFullRuntime);
  else
    __kmpc_generic_kernel_deinit();
}

#ifndef FORTRAN_NO_LONGER_NEEDS
EXTERN void __kmpc_spmd_kernel_init(int ThreadLimit,
                                    int16_t RequiresOMPRuntime) {
  PRINT0(LD_IO, "call to __kmpc_spmd_kernel_init\n");

  setExecutionParameters(OMP_TGT_EXEC_MODE_SPMD,
		         RequiresOMPRuntime ? OMP_TGT_RUNTIME_INITIALIZED
                                            : OMP_TGT_RUNTIME_UNINITIALIZED);
  int threadId = __kmpc_get_hardware_thread_id_in_block();
  if (threadId == 0) {
    usedSlotIdx = __kmpc_impl_smid() % MAX_SM;
    parallelLevel[0] =
        1 + (__kmpc_get_hardware_num_threads_in_block() > 1 ? OMP_ACTIVE_PARALLEL_LEVEL : 0);
  } else if (GetLaneId() == 0) {
    parallelLevel[GetWarpId()] =
        1 + (__kmpc_get_hardware_num_threads_in_block() > 1 ? OMP_ACTIVE_PARALLEL_LEVEL : 0);
  }
  if (!RequiresOMPRuntime) {
    // Runtime is not required - exit.
    __kmpc_impl_syncthreads();
    return;
  }

  //
  // Team Context Initialization.
  //
  // In SPMD mode there is no master thread so use any cuda thread for team
  // context initialization.
  if (threadId == 0) {
    // Get a state object from the queue.
    omptarget_nvptx_threadPrivateContext =
        omptarget_nvptx_device_State[usedSlotIdx].Dequeue();

    omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
    omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();
    // init team context
    currTeamDescr.InitTeamDescr();
#ifdef OMPD_SUPPORT
    ompd_init();
    ompd_bp_parallel_begin(); // This should be placed later, but the parallel
                              // handle is ready from here on.
#endif /*OMPD_SUPPORT*/
  }
  __kmpc_impl_syncthreads();

  omptarget_nvptx_TeamDescr &currTeamDescr = getMyTeamDescriptor();
  omptarget_nvptx_WorkDescr &workDescr = getMyWorkDescriptor();

  //
  // Initialize task descr for each thread.
  //
  omptarget_nvptx_TaskDescr *newTaskDescr =
      omptarget_nvptx_threadPrivateContext->Level1TaskDescr(threadId);
  ASSERT0(LT_FUSSY, newTaskDescr, "expected a task descr");
  newTaskDescr->InitLevelOneTaskDescr(currTeamDescr.LevelZeroTaskDescr());
  // install new top descriptor
  omptarget_nvptx_threadPrivateContext->SetTopLevelTaskDescr(threadId,
                                                             newTaskDescr);

  // init thread private from init value
  PRINT(LD_PAR,
        "thread will execute parallel region with id %d in a team of "
        "%d threads\n",
        (int)newTaskDescr->ThreadId(), (int)ThreadLimit);

#ifdef OMPD_SUPPORT
  ompd_init_thread_parallel(); // __kmpc_kernel_parallel() is not called in
                               // spmd mode
  ompd_bp_thread_begin();
#endif
}
EXTERN void __kmpc_spmd_kernel_deinit_v2(int16_t RequiresOMPRuntime) {
  // We're not going to pop the task descr stack of each thread since
  // there are no more parallel regions in SPMD mode.
  if (!RequiresOMPRuntime)
    return;

  __kmpc_impl_syncthreads();
#ifdef OMPD_SUPPORT
  ompd_bp_thread_end();
#endif
  int threadId = __kmpc_get_hardware_thread_id_in_block();
  if (threadId == 0) {
#ifdef OMPD_SUPPORT
    ompd_bp_parallel_end();
#endif
    // Enqueue omp state object for use by another team.
    int slot = usedSlotIdx;
    omptarget_nvptx_device_State[slot].Enqueue(
        omptarget_nvptx_threadPrivateContext);
  }
}
#endif // FORTRAN_NO_LONGER_NEEDS

#pragma omp end declare target
